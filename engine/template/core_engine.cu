#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <hip/hip_runtime_api.h>
#include <>

#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>


#include "generated_core_engine.cuh"

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)


//#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

//template<typename T>
//void check(T err, const char* const func, const char* const file, const int line) {
//  if (err != hipSuccess) {
//    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
//    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
//    exit(1);
//  }
//}

__global__ void update_positions_kernel(
		float *d_p_x_out,
		float *d_p_y_out,
		float delta_t,
		float *d_p_x,
		float *d_p_y,
		float *d_nu
		)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	d_p_x_out[idx] = p_x_func[idx](delta_t, d_p_x, d_p_y, d_nu);
	d_p_y_out[idx] = p_y_func[idx](delta_t, d_p_x, d_p_y, d_nu);
}


__global__ void update_atoms_quantities_kernel(
		float *d_nu_out,
		float delta_t,
		float *d_p_x,
		float *d_p_y,
		float *d_nu)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  d_nu_out[idx] = nu_func[idx](delta_t, d_p_x, d_p_y, d_nu);
}

__global__ void split_xyzw_to_x_and_y(float *d_p_x_out, float *d_p_y_out, float4 *d_xyzw_position_array)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  d_p_x_out[idx] = d_xyzw_position_array[idx].x;
  d_p_y_out[idx] = d_xyzw_position_array[idx].y;
}

__global__ void join_x_y_to_xyzw(float4 *d_xyzw_position_out, float *d_p_x, float *d_p_y)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  d_xyzw_position_out[idx].x = d_p_x[idx];
  d_xyzw_position_out[idx].y = d_p_y[idx];
}

void allocate_arrays(float * &p_x,
		     float * &p_y,
		     float * &nu,
		     float * &d_p_x,
		     float * &d_p_y,
		     float * &d_nu,
		     float * &d_p_x_out,
		     float * &d_p_y_out,
		     float * &d_nu_out)
{
  p_x = new float[NUMBER_OF_MATTERS];
  p_y = new float[NUMBER_OF_MATTERS];
  nu = new float[NUMBER_OF_MATTERS * NUMBER_OF_ATOMS];

  CUDA_CHECK_RETURN(hipMalloc((void **) &d_p_x, sizeof(float) * NUMBER_OF_MATTERS));
  CUDA_CHECK_RETURN(hipMalloc((void **) &d_p_y, sizeof(float) * NUMBER_OF_MATTERS));
  CUDA_CHECK_RETURN(hipMalloc((void **) &d_nu, sizeof(float) * NUMBER_OF_MATTERS * NUMBER_OF_ATOMS));

  CUDA_CHECK_RETURN(hipMalloc((void **) &d_p_x_out, sizeof(float) * NUMBER_OF_MATTERS));
  CUDA_CHECK_RETURN(hipMalloc((void **) &d_p_y_out, sizeof(float) * NUMBER_OF_MATTERS));
  CUDA_CHECK_RETURN(hipMalloc((void **) &d_nu_out, sizeof(float) * NUMBER_OF_MATTERS * NUMBER_OF_ATOMS));
}

void free_arrays(float * &p_x,
                 float * &p_y,
                 float * &nu,
                 float * &d_p_x,
                 float * &d_p_y,
                 float * &d_nu,
                 float * &d_p_x_out,
                 float * &d_p_y_out,
                 float * &d_nu_out)
{
  CUDA_CHECK_RETURN(hipFree(d_p_x));
  CUDA_CHECK_RETURN(hipFree(d_p_y));
  CUDA_CHECK_RETURN(hipFree(d_nu));

  CUDA_CHECK_RETURN(hipFree(d_p_x_out));
  CUDA_CHECK_RETURN(hipFree(d_p_y_out));
  CUDA_CHECK_RETURN(hipFree(d_nu_out));

  delete[] p_x;
  delete[] p_y;
  delete[] nu;
}

void upload_host_arrays(float *d_p_x,
			float *d_p_y,
			float *d_nu,
			float *p_x,
			float *p_y,
			float *nu)
{
  CUDA_CHECK_RETURN(hipMemcpy(d_p_x, p_x, sizeof(float) * NUMBER_OF_MATTERS, hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(d_p_y, p_y, sizeof(float) * NUMBER_OF_MATTERS, hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(d_nu, nu, sizeof(float) * NUMBER_OF_MATTERS * NUMBER_OF_ATOMS, hipMemcpyHostToDevice));
}

void join_position_arrays(float4 *d_xyzw_p,
			  float *d_p_x,
			  float *d_p_y)
{
  const dim3 blockSize(NUMBER_OF_MATTERS, 1, 1);
  const dim3 gridSize( 1, 1, 1);

  join_x_y_to_xyzw<<<gridSize, blockSize>>>(d_xyzw_p, d_p_x, d_p_y);
}

void integrate(float delta_t,
               float4 *d_xyzw_p,
               float *&d_nu,
               float *&d_nu_out,
               float *&d_p_x,
               float *&d_p_y,
               float *&d_p_x_out,
               float *&d_p_y_out)
{
  // I believe that blockSize should be as large as possible
  // so we should determine that value by querying GPU
  // deviceProp.maxThreadsPerBlock
  
  const dim3 blockSize(NUMBER_OF_MATTERS, 1, 1);
  const dim3 gridSize( 1, 1, 1);
  const dim3 blockSizeAtoms(NUMBER_OF_MATTERS * NUMBER_OF_ATOMS, 1, 1);

  split_xyzw_to_x_and_y<<<gridSize, blockSize>>>(d_p_x, d_p_y, d_xyzw_p);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	
  update_positions_kernel<<<gridSize, blockSize>>>(d_p_x_out, d_p_y_out, delta_t, d_p_x, d_p_y, d_nu);  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  update_atoms_quantities_kernel<<<gridSize, blockSizeAtoms>>>(d_nu_out, delta_t, d_p_x, d_p_y, d_nu);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  join_x_y_to_xyzw<<<gridSize, blockSize>>>(d_xyzw_p, d_p_x_out, d_p_y_out);

  float *tmp_nu = d_nu;
  d_nu = d_nu_out;
  d_nu_out = tmp_nu;

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

void cudaGLInit(int argc, char **argv)
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaGLDevice(argc, (const char **)argv);
}

void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{
    checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
                                                 cudaGraphicsMapFlagsNone));
}

void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
}

void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
    void *ptr;
    checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,
                                                         *cuda_vbo_resource));
    return ptr;
}

void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
    checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
    if (err == hipSuccess)
        return;
    std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
    exit (1);
}